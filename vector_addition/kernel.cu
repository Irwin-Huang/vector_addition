#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#define TPB 128

__global__ 
void addKernel(int *d_c, int *d_a, int *d_b)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
    d_c[i] = d_a[i] + d_b[i];
	printf("%u plus %u is %u.\n", d_a[i], d_b[i], d_c[i]);
}

void addArray(int *c, int *a, int *b, int len)
{
	int *d_a = 0;
	int *d_b = 0;
	int *d_c = 0;

	hipMalloc(&d_a, len * sizeof(int));
	hipMalloc(&d_b, len * sizeof(int));
	hipMalloc(&d_c, len * sizeof(int));

	hipMemcpy(d_a, a, len * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, len * sizeof(int), hipMemcpyHostToDevice);

	addKernel << <len / TPB, TPB >> > (d_c, d_a, d_b);
	hipMemcpy(c, d_c, len * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

}

